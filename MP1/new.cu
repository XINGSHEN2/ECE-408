#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int n){
    int i = threadId.x + blockDim.x * blockIdx.x;
    if(i < n) C[i] = A[i] + B[i];
}


void vecAdd(float* A, float* B, float* C, int n)
{
    int size = n* sizeof(float);
    float *d_A *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);
    hipMemcpy(C, d_C, size, cudaMencpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);
  
}