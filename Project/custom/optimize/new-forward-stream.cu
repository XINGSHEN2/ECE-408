#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16

// __constant__ float const_mask[6000];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    // #define shared_4d(i2, i1, i0) shared[(i2) * (shared_width * shared_width) + (i1) * (shared_width) + i0]

    // Insert your GPU convolution kernel code here
    int W_grid = ceil(1.0 * W_out / TILE_WIDTH);
    int b = blockIdx.z;
    // int w_base = TILE_WIDTH * (b % W_grid);
    // int h_base = TILE_WIDTH * (b/ W_grid)
    int m = blockIdx.x;
    int h = (blockIdx.y / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_grid) * TILE_WIDTH + threadIdx.x;

    float acc = 0.0f;
    

    for(int c = 0; c < C; c++){
        for (int p = 0; p < K; p++)
            for (int q = 0; q < K; q++)
                acc += in_4d(b, c, h * S + p, w * S + q) * mask_4d(m, c, p, q);
        }
    if(h < H_out && w < W_out){
        out_4d(b, m, h, w) = acc;
    }



    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
#define STREAM_NUM 1
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    float* host_out_temp = (float*) host_output;

    int input_size = B * C * H * W / STREAM_NUM;
    int output_size = B * M * H_out * W_out / STREAM_NUM;
    int mask_size = M * C * K * K;

    int W_grid = (W_out + TILE_WIDTH - 1) / TILE_WIDTH;
    int H_grid = (H_out + TILE_WIDTH - 1) / TILE_WIDTH;
    int Y_grid = H_grid * W_grid;

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDim(M, Y_grid, B/STREAM_NUM);

    hipMalloc((void **) device_input_ptr, B * C * H * W * sizeof(float));
    hipMalloc((void **) device_output_ptr, B * M * H_out * W_out * sizeof(float));
    hipMalloc((void **) device_mask_ptr, mask_size * sizeof(float));

    hipStream_t stream[STREAM_NUM];
    for (int i = 0; i < STREAM_NUM; i++)
        hipStreamCreate(&stream[i]);

    hipMemcpyAsync(*device_mask_ptr, host_mask, mask_size * sizeof(float), hipMemcpyHostToDevice, stream[0]);
    for (int i = 0; i < STREAM_NUM; i++){
        int in_offset = input_size * i;
        int out_offset = output_size * i;
        hipMemcpyAsync((*device_input_ptr) + in_offset, host_input + in_offset, input_size * sizeof(float), hipMemcpyHostToDevice, stream[i]);
        conv_forward_kernel<<<gridDim, blockDim, 0, stream[i]>>>((*device_output_ptr) + out_offset, (*device_input_ptr) + in_offset, *device_mask_ptr, B, M, C, H, W, K, S);
        hipMemcpyAsync(host_out_temp + out_offset, (*device_output_ptr) + out_offset, output_size * sizeof(float), hipMemcpyDeviceToHost, stream[i]);
    }
    hipDeviceSynchronize();

    for (int i = 0; i < STREAM_NUM; i++){
        hipStreamDestroy(stream[i]);
    }

    // Free device memory
    hipFree(device_input_ptr);
    hipFree(device_output_ptr);
    hipFree(device_mask_ptr);
#undef STREAM_NUM
    // hipMemcpy(*device_input_ptr, host_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_mask_ptr, host_mask, mask_size * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpyToSymbol(HIP_SYMBOL(const_mask), host_mask, mask_size * sizeof(float));
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    // const int H_out = (H - K)/S + 1;
    // const int W_out = (W - K)/S + 1;
    // const int shared_width = K + S * (TILE_WIDTH - 1);

    // int H_grid = ceil(1.0 * H_out / TILE_WIDTH);
    // int W_grid = ceil(1.0 * W_out / TILE_WIDTH);

    // int Y_grid = H_grid * W_grid;

    // int shared_men_size = C * shared_width * shared_width * sizeof(float);

    // dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    // dim3 gridDim(B, M, Y_grid);


    // conv_forward_kernel<<<gridDim, blockDim, shared_men_size>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    return;
    

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    // const int H_out = (H - K)/S + 1;
    // const int W_out = (W - K)/S + 1;

    // int output_size = B * M * H_out * W_out;

    // hipMemcpy(host_output, device_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
   
    // // Free device memory
    // hipFree(device_input);
    // hipFree(device_output);
    // hipFree(device_mask);
    return;

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
